//------------------------------------------------------------------------------
// GraphBLAS/CUDA/GB_cuda_get_device_count.cu: find out how many GPUs exist
//------------------------------------------------------------------------------

// SuiteSparse:GraphBLAS, Timothy A. Davis, (c) 2017-2024, All Rights Reserved.
// This file: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
// SPDX-License-Identifier: Apache-2.0

//------------------------------------------------------------------------------

#include "GB_cuda.hpp"

bool GB_cuda_get_device_count   // true if OK, false if failure
(
    int *gpu_count              // return # of GPUs in the system
)
{
    hipError_t err = hipGetDeviceCount (gpu_count) ;
    return (err == hipSuccess) ;
}

