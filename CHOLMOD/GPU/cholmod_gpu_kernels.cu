#include "hip/hip_runtime.h"
/* ========================================================================== */
/* ========================= CHOLMOD CUDA/C kernels ========================= */
/* ========================================================================== */

//------------------------------------------------------------------------------
// CHOLMOD/GPU Module.  Copyright (C) 2005-2022, Timothy A. Davis.
// All Rights Reserved.
// SPDX-License-Identifier: GPL-2.0+
//------------------------------------------------------------------------------

#include <stdio.h>
#include "SuiteSparse_config.h"
#include "cholmod.h"

#if defined ( CHOLMOD_HAS_CUDA )

/* 64-bit version only */

extern "C" {

  __global__ void kernelCreateMap ( int64_t *d_Map, int64_t *d_Ls, 
				    int64_t psi, int64_t nsrow )
  /*
    Ls[supernode row] = Matrix Row
  */
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < nsrow ) {
      d_Map[d_Ls[psi+tid]] = ((int64_t) (tid));
    }
  }
  
  __global__ void kernelCreateRelativeMap ( int64_t *d_Map, int64_t *d_Ls, 
					    int64_t *d_RelativeMap, 
					    int64_t pdi1, int64_t ndrow )
  {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if ( tid < ndrow ) {
      d_RelativeMap[tid] = d_Map[d_Ls[pdi1+tid]];
    }
  }
  
  __global__ void kernelAddUpdate ( double *d_A, double *devPtrC, 
				    int64_t *d_RelativeMap, 
				    int64_t ndrow1, int64_t ndrow2, 
				    int64_t nsrow )
  {
    int idrow = blockIdx.x * blockDim.x + threadIdx.x;
    int idcol = blockIdx.y * blockDim.y + threadIdx.y;
    if ( idrow < ndrow2  && idcol < ndrow1 ) {
      int64_t idx = d_RelativeMap[idrow] + d_RelativeMap[idcol] * nsrow;
      d_A[idx] += devPtrC[idrow+ndrow2*idcol];
    }
  }
  
  __global__ void kernelAddComplexUpdate ( double *d_A, double *devPtrC, 
					   int64_t *d_RelativeMap, 
					   int64_t ndrow1, int64_t ndrow2, 
					   int64_t nsrow )
  {
    int idrow = blockIdx.x * blockDim.x + threadIdx.x;
    int idcol = blockIdx.y * blockDim.y + threadIdx.y;
    if ( idrow < ndrow2  && idcol < ndrow1 ) {
      int64_t idx = d_RelativeMap[idrow] + d_RelativeMap[idcol] * nsrow;
      d_A[idx*2] += devPtrC[(idrow+ndrow2*idcol)*2];
      d_A[idx*2+1] += devPtrC[(idrow+ndrow2*idcol)*2+1];
    }
  }
  
  __global__ void kernelSumA ( double *a1, double *a2, const double alpha, 
			       int nsrow, int nscol ) {
    int isrow = blockIdx.x * blockDim.x + threadIdx.x;
    int iscol = blockIdx.y * blockDim.y + threadIdx.y;
    if ( isrow < nsrow && iscol < nscol ) {
      int64_t idx = iscol*nsrow + isrow;
      a1[idx] += alpha * a2[idx];
    }
  }

  __global__ void kernelSumComplexA ( double *a1, double *a2, 
				      const double alpha, int nsrow, 
				      int nscol ) {
    int isrow = blockIdx.x * blockDim.x + threadIdx.x;
    int iscol = blockIdx.y * blockDim.y + threadIdx.y;
    if ( isrow < nsrow && iscol < nscol ) {
      int64_t idx = iscol*nsrow + isrow;
      a1[idx*2] += alpha * a2[idx*2];
      a1[idx*2+1] += alpha * a2[idx*2+1];
    }
  }

  /* ======================================================================== */
  /* using Ls and Lpi data already on the device, construct Map */
  /* ======================================================================== */
  int createMapOnDevice ( int64_t *d_Map, int64_t *d_Ls, 
			  int64_t  psi, int64_t nsrow ) 
  {
    unsigned int kgrid = (nsrow+31)/32;
    unsigned int kblock = 32;
    kernelCreateMap <<<kgrid, kblock>>> ( d_Map, d_Ls, psi, nsrow );
    return 0;
  }


  int createRelativeMapOnDevice ( int64_t *d_Map, int64_t *d_Ls, 
				  int64_t *d_RelativeMap,int64_t  pdi1, 
				  int64_t ndrow, hipStream_t* astream )
  {
    unsigned int kgrid = (ndrow+255)/256;
    unsigned int kblock = 256;
    kernelCreateRelativeMap <<<kgrid, kblock, 0, *astream>>> 
      ( d_Map, d_Ls, d_RelativeMap, pdi1, ndrow);
    return 0;
  }


  /* ======================================================================== */
  int addUpdateOnDevice ( double *d_A, double *devPtrC, 
			  int64_t *d_RelativeMap, int64_t ndrow1, 
			  int64_t ndrow2, int64_t nsrow, 
			  hipStream_t* astream )
  /* ======================================================================== */
  /* Assemble the Schur complment from a descendant supernode into the current
     supernode */ 
  /* ======================================================================== */
{
  dim3 grids;
  dim3 blocks;

  blocks.x = 16;
  blocks.y = 16;
  blocks.z = 1;

  grids.x = (ndrow2+15)/16; 
  grids.y = (ndrow1+15)/16; 

  kernelAddUpdate <<<grids, blocks, 0, *astream>>> 
    ( d_A, devPtrC, d_RelativeMap, ndrow1, ndrow2, nsrow );

  return 0;
}

  /* ======================================================================== */
  int addComplexUpdateOnDevice ( double *d_A, double *devPtrC, 
				 int64_t *d_RelativeMap, int64_t ndrow1, 
				 int64_t ndrow2, int64_t nsrow, 
				 hipStream_t* astream )
  /* ======================================================================== */
  /* Assemble the Schur complment from a descendant supernode into the current
     supernode */ 
  /* ======================================================================== */
{
  dim3 grids;
  dim3 blocks;

  blocks.x = 16;
  blocks.y = 16;
  blocks.z = 1;

  grids.x = (ndrow2+15)/16; 
  grids.y = (ndrow1+15)/16; 

  kernelAddComplexUpdate <<<grids, blocks, 0, *astream>>> 
    ( d_A, devPtrC, d_RelativeMap, ndrow1, ndrow2, nsrow );

  return 0;
}

  int sumAOnDevice ( double *a1, double *a2, const double alpha, 
		     int nsrow, int nscol )
  {
    dim3 grids;
    dim3 blocks;
    blocks.x = 16;
    blocks.y = 16;
    blocks.z = 1;
    grids.x = (nsrow+15)/16;
    grids.y = (nscol+15)/16;
    kernelSumA <<<grids, blocks, 0, 0>>> ( a1, a2, alpha, nsrow, nscol );
    return 0;
  }

  int sumComplexAOnDevice ( double *a1, double *a2, const double alpha, 
			    int nsrow, int nscol )
  {
    dim3 grids;
    dim3 blocks;
    blocks.x = 16;
    blocks.y = 16;
    blocks.z = 1;
    grids.x = (nsrow+15)/16;
    grids.y = (nscol+15)/16;
    kernelSumComplexA <<<grids, blocks, 0, 0>>> ( a1, a2, alpha, nsrow, nscol );
    return 0;
  }

}

#endif
